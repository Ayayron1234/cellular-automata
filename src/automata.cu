#include "hip/hip_runtime.h"
﻿#include "automata.h"

#include "utils/ExternalResource.h"
#include "IO.h"

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__device__ Float maxf(Float a, Float b) {
    return (a > b) ? a : b;
}

__device__ Float minf(Float a, Float b) {
    return (a < b) ? a : b;
}

__device__ int generateRandomInt(int thread_id, int seed) {
    static hiprandState* state = nullptr;
    if (state == nullptr) {
        state = new hiprandState{};
        hiprand_init(seed, thread_id, 0, state);
    }
    return hiprand(state);
}

/**
 * CUDA kernel function: Computes the color of a pixel in the Mandelbrot or Julia set and stores it in the buffer.
 *
 * @param buffer - The output buffer storing RGB values of pixels.
 * @param options - The Mandelbrot set properties and camera configuration.
 * @param maxIterations - The maximum number of iterations for the fractal computation.
 */
__global__ void calcPixel(IO::RGB* buffer, Options options, const ConwayGrid grid) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = row * options.windowWidth + col;

    Float wph = (Float)options.windowWidth / (Float)options.windowHeight;
    Float x0 = ((Float)col / (Float)options.windowWidth) / options.camera.zoom - (0.5 / options.camera.zoom) - options.camera.position.x;
    Float y0 = (((Float)row / (Float)options.windowHeight) / options.camera.zoom - (0.5 / options.camera.zoom) - options.camera.position.y) / wph;

    int x = floor(x0), y = floor(y0);

    const int s = 1;
    float sum_neighborhood = 0;
    for (int d = -12; d <= 12; ++d)
        sum_neighborhood += grid.get(x + d % 5, y + d / 5) * (5 - (abs(d % 5) + abs(d / 5)));
    //int sum_neighborhood = grid.get(x + 1, y) + grid.get(x + 1, y + 1) + grid.get(x, y + 1) + grid.get(x - 1, y + 1)
    //    + grid.get(x - 1, y) + grid.get(x - 1, y - 1) + grid.get(x, y - 1) + grid.get(x + 1, y - 1);

    char color = 255.f * sum_neighborhood / (float)25;
        
    // Store the RGB color values in the buffer, clamped to the range [0, 255]
    buffer[i].r = minf(255.f, color);
    buffer[i].g = minf(255.f, color);
    buffer[i].b = minf(255.f, color);
}

template <typename T>
__global__ void calcNextState(Options options, const ConwayGrid grid) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int i = y * grid.width() + x;

    T cell = grid.get(x, y);
    T newCell = grid.outerValue;
    int sum_neighborhood = grid.get(x + 1, y) + grid.get(x + 1, y + 1) + grid.get(x, y + 1) + grid.get(x - 1, y + 1)
        + grid.get(x - 1, y) + grid.get(x - 1, y - 1) + grid.get(x, y - 1) + grid.get(x + 1, y - 1);
    if (cell == 1) {
        if (sum_neighborhood < 2)
            newCell = 0;
        else if (sum_neighborhood < 4)
            newCell = 1;
        else
            newCell = 0;
    }
    else {
        if (sum_neighborhood == 3)
            newCell = 1;
    }

    if (generateRandomInt(i, 0) % 5000 == 0)
        newCell = 1;

    grid.set(x, y, newCell);
}

hipError_t conwayCuda(Options options, ConwayGrid* _grid, bool advanceState) {
    using cell_t = std::remove_pointer_t<decltype(_grid)>::cell_t;

    IO::RGB* gpuBuffer = 0;
    hipError_t cudaStatus;
    std::remove_pointer_t<decltype(_grid)> grid = *_grid;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&gpuBuffer, options.windowWidth * options.windowHeight * sizeof(IO::RGB));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    grid.setCudaBuffer();

    dim3 block_size(20, 20);

    dim3 grid_size(grid.width() / block_size.x, grid.height() / block_size.y);
    if (advanceState)
        calcNextState<cell_t> <<< grid_size, block_size >>> (options, grid);

    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}

    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //    goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy((unsigned char*)grid->getInputBuffer(), gridGpuBuffer, grid->width() * grid->height() * sizeof(cell_t), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}

    grid_size = dim3(options.windowWidth / block_size.x, options.windowHeight / block_size.y);
    calcPixel <<< grid_size, block_size >>> (gpuBuffer, options, grid);

    grid.loadCudaBuffer();

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Cleanup;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Cleanup;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy((unsigned char*)IO::GetOutputBuffer(), gpuBuffer, options.windowWidth * options.windowHeight * sizeof(IO::RGB), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        goto Cleanup;
    }

Cleanup:
    hipFree(gpuBuffer);
    grid.cleanupCudaBuffer();

    //std::cout << "done" << std::endl;

    return cudaStatus;
}
