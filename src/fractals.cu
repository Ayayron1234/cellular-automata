#include "hip/hip_runtime.h"
﻿#include "utils/Json.h"
#include "utils/ExternalResource.h"
#include "IO.h"

#include <stdio.h>

JSON_C(vec2, JSON_M(x), JSON_M(y))

struct Camera {
    vec2 position;
    Float zoom;
}; JSON_C(Camera, JSON_M(position), JSON_M(zoom))

enum class FractalType { Mandelbrot = 0x00, Julia, _COUNT };

struct Options {
    int windowWidth, windowHeight;
    Camera camera{ vec2(), 0.2 };

    FractalType type = FractalType::Julia;
    int baseIterations = 50;
    Float iterationIncreaseFallOff = 12.l;
    vec2 z0;
    vec2 c;

    void SetProperty(vec2 value) {
        switch (type)
        {
        case FractalType::Mandelbrot:
            c = value; break;
        case FractalType::Julia:
            z0 = value; break;
        default:
            break;
        }
    }

    __device__ __host__ vec2 GetProperty() {
        if (type == FractalType::Mandelbrot) return c;
        return z0;
    }

}; JSON_C(Options, JSON_M(windowWidth), JSON_M(windowHeight), JSON_M(camera), JSON_M(type), JSON_M(z0), JSON_M(c))

__device__ vec2 calcNext(vec2 z, vec2 c) {
    const Float zr = z.x * z.x - z.y * z.y;
    const Float zi = 2.f * z.x * z.y;

    return vec2{ zr, zi } + c;
}

__device__ int calcIterations(vec2 z0, vec2 c, int max_iter) {
    vec2 zn = z0;
    int iter = 0;

    while ((zn.x * zn.x + zn.y * zn.y <= 4.0f) && (iter < max_iter)) {
        zn = calcNext(zn, c);
        iter++;
    }

    return iter;
}

__device__ Float maxf(Float a, Float b) {
    return (a > b) ? a : b;
}

__device__ Float minf(Float a, Float b) {
    return (a < b) ? a : b;
}

__global__ void calcMandelbrot(IO::RGB* buffer, Options options, int maxIterations) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = row * options.windowWidth + col;

    Float wph = (Float)options.windowWidth / (Float)options.windowHeight;
    Float x0 = ((Float)col / (Float)options.windowWidth) / options.camera.zoom - (0.5 / options.camera.zoom) - options.camera.position.x;
    Float y0 = (((Float)row / (Float)options.windowHeight) / options.camera.zoom - (0.5 / options.camera.zoom) - options.camera.position.y) / wph;

    vec2 z, c;
    if (options.type == FractalType::Mandelbrot) {
        z = options.GetProperty();
        c = { x0, y0 };
    }
    else {
        z = { x0, y0 };
        c = options.GetProperty();
    }

    int iter = 0;
    Float xtemp = 0;
    while ((z.x * z.x + z.y * z.y <= 4.0f) && (iter < maxIterations)) {
        xtemp = z.x * z.x - z.y * z.y + c.x;
        z.y = 2.0f * z.x * z.y + c.y;
        z.x = xtemp;
        iter++;
    }

    Float color = 
        //5.0 * 
        ((Float)iter 
         * 255.l / (Float)maxIterations
         -  log2f(maxf(1.f, log2f(length(z))))
        );


    buffer[i].r = minf(255.f, color);
    buffer[i].g = minf(255.f, color);
    buffer[i].b = minf(255.f, color);
}

hipError_t mandelbrotCuda(Options options, int maxIterations) {
    IO::RGB* gpuBuffer = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&gpuBuffer, options.windowWidth * options.windowHeight * sizeof(IO::RGB));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    dim3 block_size(16, 16);
    dim3 grid_size(options.windowWidth / block_size.x, options.windowHeight / block_size.y);

    calcMandelbrot << < grid_size, block_size >> > (gpuBuffer, options, maxIterations);

    //// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy((unsigned char*)IO::GetOutputBuffer(), gpuBuffer, options.windowWidth * options.windowHeight * sizeof(IO::RGB), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(gpuBuffer);

    return cudaStatus;
}

auto& g_options = external_resource<"options.json", Json::wrap<Options>>::value;

int main() {
    IO::OpenWindow(g_options.windowWidth, g_options.windowHeight);
    
    vec2 dragStart; // normalized
    while (!SDL_QuitRequested()) {
        IO::HandleEvents();
        g_options.windowWidth = IO::GetWindowWidth();
        g_options.windowHeight = IO::GetWindowHeight();

        vec2 normalizedMousePos = IO::NormalizePixel(IO::GetMousePos().x, IO::GetMousePos().y);

        const Uint8* state = SDL_GetKeyboardState(nullptr);
        if (state[SDL_SCANCODE_SPACE]) {
            g_options.SetProperty({ 0, 0 });
            g_options.camera.position = { 0, 0 };
            g_options.camera.zoom = 0.2;
        }

        static vec2 z0Start;
        if (IO::MouseClicked(SDL_BUTTON_LEFT)) {
            z0Start = g_options.GetProperty();
            dragStart = normalizedMousePos;
        }
        if (IO::IsButtonDown(SDL_BUTTON_LEFT)) {
            g_options.SetProperty(z0Start + 0.25l * (normalizedMousePos - dragStart) / g_options.camera.zoom);
        }

        static Float zoom = 1.l;
        static Float zoomDP = 1.02;
        static Float zoomDN = 1.035;
        if (IO::GetMouseWheel() > 0) 
            zoom = zoomDP;
        else if (IO::GetMouseWheel() < 0)
            zoom = 1.l / zoomDN;
        if (abs(zoom - 1.l) > 0.0001f)
            g_options.camera.position = g_options.camera.position - 0.5l * normalizedMousePos / g_options.camera.zoom + 0.5l * normalizedMousePos / (g_options.camera.zoom * zoom);
        g_options.camera.zoom *= zoom;
        zoom = 1.l + (zoom - 1.l) * 0.975l;

        mandelbrotCuda(g_options, (float)g_options.baseIterations * powl(g_options.camera.zoom, 1.0l / g_options.iterationIncreaseFallOff));
        
    	IO::Render();
    }
    
    IO::Quit();

    return 0;
}
