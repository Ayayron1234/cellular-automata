#include "hip/hip_runtime.h"
#include "../kernels.h"   
#include <hiprand/hiprand_kernel.h>
#include "../utils/IO.h"

__global__
void setPixel(GlobalBuffer<IO::RGB> pixelBuffer, Options options, Grid<Cell> grid) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = row * options.windowWidth + col;

    Float wph = (Float)options.windowWidth / (Float)options.windowHeight;
    Float x0 = ((Float)col / (Float)options.windowWidth) / options.camera.zoom - (0.5 / options.camera.zoom) - options.camera.position.x;
    Float y0 = (((Float)row / (Float)options.windowHeight) / options.camera.zoom - (0.5 / options.camera.zoom) - options.camera.position.y) / wph;

    int x = floor(x0), y = floor(y0);

    unsigned int val = grid.Get(x, y, {false, 1}).type;
    IO::RGB color;
    if (val == 1)
        color = { (char)191, (char)174, (char)111 };
    else if (val == 2) {
        //if (grid.Get(x + 1, y).type == 0 || grid.Get(x - 1, y).type == 0)
        //    color = IO::RGB();
        //else if (grid.Get(x, y - 1).type == 0)
        //    color = IO::RGB( 98, 152, 163 );
        //else
            color = IO::RGB(54, 124, 138 );
    }

    pixelBuffer.Write(i, color);
}

void drawGrid(GlobalBuffer<IO::RGB> pixelBuffer, Grid<Cell> grid, Options options) {
    dim3 block_size(32, 32);
    dim3 grid_size(options.windowWidth / block_size.x, options.windowHeight / block_size.y);

    setPixel << < grid_size, block_size >> > (pixelBuffer, options, grid);
}
